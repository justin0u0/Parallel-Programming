// Author: justin0u0<mail@justin0u0.com>
//
// Finished basic blocked floyd warshall algorithm with no optimization


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int BLOCK_SIZE = 32;

void handleInput(const char* inputFile, int& n, int& m, int** hostD) {
	FILE* file = fopen(inputFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	*hostD = (int*)malloc(n * n * sizeof(int));

	int** d = (int**)malloc(n * sizeof(int*));
	for (int i = 0; i < n; ++i) {
		d[i] = (*hostD) + i * n;
		for (int j = 0; j < n; ++j) {
			if (i == j) {
				d[i][j] = 0;
			} else {
				d[i][j] = INF;
			}
		}
	}

	int edge[3];
	for (int i = 0; i < m; ++i) {
		fread(edge, sizeof(int), 3, file);
		d[edge[0]][edge[1]] = edge[2];
	}
	fclose(file);
}

void handleOutput(const char* outputFile, const int n, int* hostD) {
	FILE* file = fopen(outputFile, "w");
	fwrite(hostD, sizeof(int), n * n, file);
	fclose(file);
}

__global__ void naiveFloydWarshall(const int n, const int k, int* d) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= n || y >= n) return;

	int idxIJ = x * n + y;
	int idxIK = x * n + k;
	int idxKJ = k * n + y;

	/* no cache, 6 global memory access */

	/*
		if (d[idxIJ] > d[idxIK] + d[idxKJ]) {
			d[idxIJ] = d[idxIK] + d[idxKJ];
		}
	*/

	/* shared memory cached, 2 global memory access */

	__shared__ int cacheIJ[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int cacheIK[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int cacheKJ[BLOCK_SIZE][BLOCK_SIZE];
	cacheIJ[threadIdx.x][threadIdx.y] = d[idxIJ];
	cacheIK[threadIdx.x][threadIdx.y] = d[idxIK];
	cacheKJ[threadIdx.x][threadIdx.y] = d[idxKJ];
	__syncthreads();

	if (cacheIJ[threadIdx.x][threadIdx.y] > cacheIK[threadIdx.x][threadIdx.y] + cacheKJ[threadIdx.x][threadIdx.y]) {
		d[idxIJ] = cacheIK[threadIdx.x][threadIdx.y] + cacheKJ[threadIdx.x][threadIdx.y];
	}
}

__global__ void blockedFloydWarshallPhase1(int n, int blockId, int* d) {
	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;
	int idxIJ = i * n + j;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	if (i < n && j < n) {
		cacheD[y][x] = d[idxIJ];
	} else {
		cacheD[y][x] = INF;
	}
	__syncthreads();

	// compute phase 1 - dependent phase
	int newDist;

	// TODO: unroll the loop to measure performance gained

	for (int k = 0; k < BLOCK_SIZE; ++k) {
		newDist = cacheD[y][k] + cacheD[k][x];

		if (cacheD[y][x] > newDist) {
			cacheD[y][x] = newDist;
		}

		__syncthreads();
	}

	// load shared memory back to the global memory
	if (i < n && j < n) {
		d[idxIJ] = cacheD[y][x];
	}
}

__global__ void blockedFloydWarshallPhase2(int n, int blockId, int* d) {
	// skipping the base block (from phase 1)
	if (blockIdx.x == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// blockIdx.y: [0, 1]
	// isRow is true if the block has same index i with the base block
	bool isRow = (blockIdx.y == 0);

	// load the base block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;
	int idxIJ = i * n + j;

	__shared__ int cacheBaseD[BLOCK_SIZE][BLOCK_SIZE];
	if (i < n && j < n) {
		cacheBaseD[y][x] = d[idxIJ];
	} else {
		cacheBaseD[y][x] = INF;
	}

	// load the block into shared memory
	if (isRow) {
		j = x + blockIdx.x * BLOCK_SIZE;
	} else {
		i = y + blockIdx.x * BLOCK_SIZE;
	}
	idxIJ = i * n + j;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	if (i < n && j < n) {
		cacheD[y][x] = d[idxIJ];
	} else {
		cacheD[y][x] = INF;
	}
	__syncthreads();

	// compute phase 2 - partial dependent phase
	int newDist;

	if (isRow) {
		// TODO: unroll the loop to measure performance gained

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			newDist = cacheBaseD[y][k] + cacheD[k][x];
			if (cacheD[y][x] > newDist) {
				cacheD[y][x] = newDist;
			}

			__syncthreads();
		}
	} else {
		// TODO: unroll the loop to measure performance gained

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			newDist = cacheD[y][k] + cacheBaseD[k][x];
			if (cacheD[y][x] > newDist) {
				cacheD[y][x] = newDist;
			}

			__syncthreads();
		}
	}

	// load shared memory back to the global memory
	if (i < n && j < n) {
		d[idxIJ] = cacheD[y][x];
	}
}

__global__ void blockedFloydWarshallPhase3(int n, int blockId, int* d) {
	// skipping the base blocks (from phase 1, 2)
	if (blockIdx.x == blockId || blockIdx.y == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	int i = y + blockIdx.y * BLOCK_SIZE;
	int j = x + blockIdx.x * BLOCK_SIZE;
	int idxIJ = i * n + j;

	// load the base column block (same row) into shared memory
	__shared__ int cacheBaseColD[BLOCK_SIZE][BLOCK_SIZE];
	int baseJ = x + blockId * BLOCK_SIZE;
	if (i < n && baseJ < n) {
		cacheBaseColD[y][x] = d[i * n + baseJ];
	} else {
		cacheBaseColD[y][x] = INF;
	}

	// load the base row block (same column) into shared memory
	__shared__ int cacheBaseRowD[BLOCK_SIZE][BLOCK_SIZE];
	int baseI = y + blockId * BLOCK_SIZE;
	if (baseI < n && j < n) {
		cacheBaseRowD[y][x] = d[baseI * n + j];
	} else {
		cacheBaseRowD[y][x] = INF;
	}
	__syncthreads();

	// compute phase 3 - independence phase

	if (i < n && j < n) {
		int newDist;
		int curDist = d[idxIJ];

		// TODO: unroll the loop to measure performance gained

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			newDist = cacheBaseColD[y][k] + cacheBaseRowD[k][x];
			if (curDist > newDist) {
				curDist = newDist;
			}
		}

		// load new distance back to the global memory
		d[idxIJ] = curDist;
	}
}

int main(int argc, char** argv) {
	int n, m;

	int* hostD;
	handleInput(argv[1], n, m, &hostD);

	int* deviceD;
	hipMalloc((void**)&deviceD, n * n * sizeof(int));
	
	hipMemcpy(deviceD, hostD, n * n * sizeof(int), hipMemcpyHostToDevice);

	/* naive floyd warshall */

	/*
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numberOfBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
	printf("(%d, %d) (%d, %d)\n",
		threadsPerBlock.x, threadsPerBlock.y,
		numberOfBlocks.x, numberOfBlocks.y);
	for (int k = 0; k < n; ++k) {
		naiveFloydWarshall<<<numberOfBlocks, threadsPerBlock>>>(n, k, deviceD);
	}
	*/

	/* blocked floyd warshall */

	// number of blocks is numberOfBlocks * numberOfBlocks
	int numberOfBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 gridPhase1(1, 1);
	dim3 gridPhase2(numberOfBlocks, 2); // the 2 represents the row & the column respectively
	dim3 gridPhase3(numberOfBlocks, numberOfBlocks);
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	for (int blockId = 0; blockId < numberOfBlocks; ++blockId) {
		blockedFloydWarshallPhase1<<<gridPhase1, threadsPerBlock>>>(n, blockId, deviceD);
		blockedFloydWarshallPhase2<<<gridPhase2, threadsPerBlock>>>(n, blockId, deviceD);
		blockedFloydWarshallPhase3<<<gridPhase3, threadsPerBlock>>>(n, blockId, deviceD);
	}

	hipMemcpy(hostD, deviceD, n * n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(deviceD);

	handleOutput(argv[2], n, hostD);

	return 0;
}
